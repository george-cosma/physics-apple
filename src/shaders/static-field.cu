
#include <hip/hip_runtime.h>
extern "C" __constant__ float G = 1.0 / 1000.0;

extern "C" __global__ void gravity(const float mass_product,
                                   const int* x2,
                                   const int* y2,
                                   const int attractors,
                                   float* out_x,
                                   float* out_y,
                                   const int width,
                                   const int height) {
    const int x1 = threadIdx.x + blockIdx.x * blockDim.x;
    const int y1 = threadIdx.y + blockIdx.y * blockDim.y;

    if (x1 >= width || y1 >= height) {
        return;
    }

    const int index = x1 + y1 * width;

    float result_x = 0.0;
    float result_y = 0.0;
    for (int i = 0; i < attractors; i++) {
        const float rx = x2[i] - x1;
        const float ry = y2[i] - y1;
        const float radius_squared = rx * rx + ry * ry;

        const float inv_radius_squared = (radius_squared != 0.0) ? 1.0 / radius_squared : 0.0;
        const float cos_alpha = rx * inv_radius_squared;
        const float sin_alpha = ry * inv_radius_squared;

        result_x += cos_alpha * G * mass_product * inv_radius_squared;
        result_y += sin_alpha * G * mass_product * inv_radius_squared;
    }

    out_x[index] = result_x;
    out_y[index] = result_y;
}