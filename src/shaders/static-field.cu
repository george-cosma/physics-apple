
#include <hip/hip_runtime.h>
extern "C" __constant__ float G = 1.0 / 1000.0;

extern "C" __global__ void gravity(const float mass_product, const int* x2, const int* y2, const int attractors, float* out_x, float* out_y) {
    const int x1 = threadIdx.x + blockIdx.x * blockDim.x;
    const int y1 = threadIdx.y + blockIdx.y * blockDim.y;

    // int blockId  = blockIdx.x + blockIdx.y * gridDim.x;
    // int threadId = blockId * (blockDim.x * blockDim.y) 
    //                 + (threadIdx.y * blockDim.x) + threadIdx.x;

    const int index = (threadIdx.x + blockIdx.x * blockDim.x) + (threadIdx.y + blockIdx.y * blockDim.y) * gridDim.x * blockDim.x;
    // if (index == -1) {
    //     printf("(%d, %d)", x1, y1);
    // }
    // printf("%d, ", index);
    // printf("%d\n", gridDim.x * blockDim.x);
    // if (blockIdx.x != 0 || blockIdx.y != 0) {
    //     return;
    // }
    // printf("Putting (%d, %d) at index %d.\n", x1, y1, index);


    for (int i = 0; i < attractors; i++) {
        const float rx = x2[i] - x1;
        const float ry = y2[i] - y1; 
        const float radius_squared = rx * rx + ry * ry;

        if (radius_squared != 0.0) {
            const float cos_alpha = rx / radius_squared;
            const float sin_alpha = ry / radius_squared;
            
            const float prev_x = out_x[index];
            const float prev_y = out_y[index];
            
            out_x[index] = prev_x + cos_alpha * (1.0 / 1000.0) * mass_product / radius_squared;
            out_y[index] = prev_y + sin_alpha * (1.0 / 1000.0) * mass_product / radius_squared;
        }
    }

    // if (out_x[index] == 0.0) {
    //     printf("Y is 0.");
    // }
}